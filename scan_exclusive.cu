#include "hip/hip_runtime.h"
#include "timer.h"

#define BLOCK_DIM 1024
#define M (1 << 20)  // 1M elements

__global__ void scan_kernel(float *input, float* output, float* partialSums, unsigned int N) {
    extern __shared__ float temp[];
    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + tid;
    
    // 加载数据到共享内存
    if (gid < N) {
        temp[tid] = input[gid];
    } else {
        temp[tid] = 0.0f;
    }
    __syncthreads();
    
    // 上扫（归约）
    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        int index = (tid + 1) * stride * 2 - 1;
        if (index < blockDim.x) {
            temp[index] += temp[index - stride];
        }
        __syncthreads();
    }
    
    // 保存块内总和并清零最后一个元素
    if (tid == 0) {
        partialSums[blockIdx.x] = temp[blockDim.x - 1];
        temp[blockDim.x - 1] = 0.0f;
    }
    __syncthreads();
    
    // 下扫
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        int index = (tid + 1) * stride * 2 - 1;
        if (index < blockDim.x) {
            float t = temp[index];
            temp[index] += temp[index - stride];
            temp[index - stride] = t;
        }
        __syncthreads();
    }
    
    // 写回结果
    if (gid < N) {
        output[gid] = temp[tid];
    }
}
__global__ void add_kernel(float* output, float* partialSums, unsigned int N) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        output[i] += partialSums[blockIdx.x];
    }
}

void scan_gpu_d(float* input_d, float* output_d, unsigned int N){
    Timer timer;

    // configurations
    const unsigned int numThreadsPerBlock = BLOCK_DIM;
    const unsigned int numElementsPerBlock = numThreadsPerBlock;
    const unsigned int numBlocks = (N + numElementsPerBlock -1 )/numElementsPerBlock;

    // allocate partial sums
    startTime(&timer);
    float *partialSums_d;
    hipMalloc((void**) &partialSums_d, numBlocks*sizeof(float));
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Partial sums allocation time");

    // call kernel
    startTime(&timer);
    scan_kernel<<<numBlocks, numThreadsPerBlock,numThreadsPerBlock * sizeof(float)>>>(input_d, output_d, partialSums_d, N);
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Kernel time");

    // scan partial sums then add
    if(numBlocks > 1){
        // scan partial sums
        scan_gpu_d(partialSums_d, partialSums_d, numBlocks);

        // Add scanned sums
        add_kernel<<<numBlocks, numThreadsPerBlock>>>(output_d, partialSums_d, N);

    }
    // Free memory
    startTime(&timer);
    hipFree(partialSums_d);
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Deallocation time");
}



void scan_cpu(const float* input, float* output, unsigned int N) {
    output[0] = 0.0f;
    for (unsigned int i = 1; i < N; ++i) {
        output[i] = output[i - 1] + input[i - 1];
    }
}

int main() {
    Timer timer;

    // 分配主机内存
    float* h_input = new float[M];
    float* h_output = new float[M];
    float* h_reference = new float[M];

    // 初始化输入数据
    std::srand(static_cast<unsigned int>(std::time(0)));
    for (unsigned int i = 0; i < M; ++i) {
        h_input[i] = static_cast<float>(std::rand() % 10);
    }

    // 分配 GPU 内存
    float* d_input;
    float* d_output;
    hipMalloc((void**)&d_input, M * sizeof(float));
    hipMalloc((void**)&d_output, M * sizeof(float));

    // 拷贝输入数据到 GPU
    startTime(&timer);
    hipMemcpy(d_input, h_input, M * sizeof(float), hipMemcpyHostToDevice);
    stopTime(&timer);
    printElapsedTime(timer, "Host to Device Memcpy");

    // 调用 GPU scan
    scan_gpu_d(d_input, d_output, M);

    // 拷贝结果回主机
    startTime(&timer);
    hipMemcpy(h_output, d_output, M * sizeof(float), hipMemcpyDeviceToHost);
    stopTime(&timer);
    printElapsedTime(timer, "Device to Host Memcpy");

    // 验证结果
    scan_cpu(h_input, h_reference, M);

    bool correct = true;
    for (unsigned int i = 0; i < M; ++i) {
        if (fabs(h_output[i] - h_reference[i]) > 1e-3f) {
            std::cerr << "Mismatch at index " << i << ": GPU = "
                      << h_output[i] << ", CPU = " << h_reference[i] << std::endl;
            correct = false;
            break;
        }
    }

    std::cout << (correct ? "Scan result correct ✅" : "Scan result incorrect ❌") << std::endl;

    // 清理资源
    delete[] h_input;
    delete[] h_output;
    delete[] h_reference;
    hipFree(d_input);
    hipFree(d_output);

    return 0;
}