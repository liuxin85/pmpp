
#include <hip/hip_runtime.h>
#include <stdio.h>
#define BLUR_SIZE 1

__global__ void blur_kernel(unsigned char* image, unsigned char* blurred, unsigned int width, unsigned int height){
    unsigned int outRow = blockIdx.y *blockDim.y + threadIdx.y;
    unsigned int outCol = blockIdx.x * blockDim.x + threadIdx.x;

    if(outRow < height && outCol < width){
        unsigned int average = 0;
        for(int inRow = outRow - BLUR_SIZE; inRow < outRow + BLUR_SIZE + 1; ++inRow){
            for(int inCol = outCol - BLUR_SIZE; inCol < outCol + BLUR_SIZE + 1; ++inCol){
                // 检查边界
             if (inRow >= 0 && inRow < height && inCol >= 0 && inCol < width){
                 average += image[inRow*width + inCol];
            }
            }

        }
        blurred[outRow*width + outCol] =(unsigned char) (average/((2*BLUR_SIZE + 1)*(2*BLUR_SIZE + 1)));

    }
}
int main()
{
    const unsigned int width  = 1080;
    const unsigned int height = 1080;
    const size_t       bytes  = width * height * sizeof(unsigned char);

    /* 1. 主机端分配并初始化图像 */
    unsigned char *h_img   = (unsigned char*)malloc(bytes);
    unsigned char *h_blur  = (unsigned char*)malloc(bytes);
    for (size_t i = 0; i < width * height; ++i)
        h_img[i] = rand() & 0xFF;

    /* 2. 设备端分配内存 */
    unsigned char *d_img, *d_blur;
    hipMalloc(&d_img,  bytes);
    hipMalloc(&d_blur, bytes);

    /* 3. 拷贝输入数据到 GPU */
    hipMemcpy(d_img, h_img, bytes, hipMemcpyHostToDevice);

    /* 4. 网格/线程块大小 */
    dim3 block(16, 16);
    dim3 grid((width + block.x - 1) / block.x,
              (height + block.y - 1) / block.y);

    /* 5. 计时并启动 kernel */
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    blur_kernel<<<grid, block>>>(d_img, d_blur, width, height);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms = 0.0f;
    hipEventElapsedTime(&ms, start, stop);
    printf("GPU blur took %.3f ms\n", ms);

    /* 6. 传回结果并简单校验 */
    hipMemcpy(h_blur, d_blur, bytes, hipMemcpyDeviceToHost);
    printf("First 10 output pixels: ");
    for (int i = 0; i < 10; ++i) printf("%u ", h_blur[i]);
    printf("\n");

    /* 7. 清理 */
    free(h_img);
    free(h_blur);
    hipFree(d_img);
    hipFree(d_blur);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}