#include "hip/hip_runtime.h"
#include "timer.h"

#define BLOCK_DIM 1024
#define M (1 << 20)  // 1M elements

__global__ void scan_kernel(float *input, float* output, float* partialSums, unsigned int N) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // double buffer method
    __shared__ float buffer1_s[BLOCK_DIM];
    __shared__ float buffer2_s[BLOCK_DIM];
    float* inBuffer_s = buffer1_s;
    float* outBuffer_s = buffer2_s;
    if(threadIdx.x == 0){
        inBuffer_s[threadIdx.x] = 0.0f;
    }else{
        inBuffer_s[threadIdx.x] = input[i - 1];
    }
    __syncthreads();

    for(unsigned int stride=1; stride <= BLOCK_DIM/2; stride*=2){
        if(threadIdx.x >=stride){
            outBuffer_s[threadIdx.x] = inBuffer_s[threadIdx.x] + inBuffer_s[threadIdx.x - stride];
        }else{
             outBuffer_s[threadIdx.x] = inBuffer_s[threadIdx.x];
        }
        __syncthreads();
        float* tmp = inBuffer_s;
        inBuffer_s = outBuffer_s;
        outBuffer_s = tmp;
    }
    if(threadIdx.x == BLOCK_DIM - 1){
        partialSums[blockIdx.x] = inBuffer_s[threadIdx.x] + input[i];
    }
    output[i] = inBuffer_s[threadIdx.x];

}
__global__ void add_kernel(float* output, float* partialSums, unsigned int N) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        output[i] += partialSums[blockIdx.x];
    }
}

void scan_gpu_d(float* input_d, float* output_d, unsigned int N){
    Timer timer;

    // configurations
    const unsigned int numThreadsPerBlock = BLOCK_DIM;
    const unsigned int numElementsPerBlock = numThreadsPerBlock;
    const unsigned int numBlocks = (N + numElementsPerBlock -1 )/numElementsPerBlock;

    // allocate partial sums
    startTime(&timer);
    float *partialSums_d;
    hipMalloc((void**) &partialSums_d, numBlocks*sizeof(float));
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Partial sums allocation time");

    // call kernel
    startTime(&timer);
    scan_kernel<<<numBlocks, numThreadsPerBlock,numThreadsPerBlock * sizeof(float)>>>(input_d, output_d, partialSums_d, N);
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Kernel time");

    // scan partial sums then add
    if(numBlocks > 1){
        // scan partial sums
        scan_gpu_d(partialSums_d, partialSums_d, numBlocks);

        // Add scanned sums
        add_kernel<<<numBlocks, numThreadsPerBlock>>>(output_d, partialSums_d, N);

    }
    // Free memory
    startTime(&timer);
    hipFree(partialSums_d);
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Deallocation time");
}



void exclusive_scan_cpu(const float* input, float* output, unsigned int N) {
    output[0] = 0.0f;
    for (unsigned int i = 1; i < N; ++i) {
        output[i] = output[i - 1] + input[i - 1];
    }
}

int main() {
    Timer timer;

    // 分配主机内存
    float* h_input = new float[M];
    float* h_output = new float[M];
    float* h_reference = new float[M];

    // 初始化输入数据
    std::srand(static_cast<unsigned int>(std::time(0)));
    for (unsigned int i = 0; i < M; ++i) {
        h_input[i] = static_cast<float>(std::rand() % 10);
    }

    // 分配 GPU 内存
    float* d_input;
    float* d_output;
    hipMalloc((void**)&d_input, M * sizeof(float));
    hipMalloc((void**)&d_output, M * sizeof(float));

    // 拷贝输入数据到 GPU
    startTime(&timer);
    hipMemcpy(d_input, h_input, M * sizeof(float), hipMemcpyHostToDevice);
    stopTime(&timer);
    printElapsedTime(timer, "Host to Device Memcpy");

    // 调用 GPU scan
    scan_gpu_d(d_input, d_output, M);

    // 拷贝结果回主机
    startTime(&timer);
    hipMemcpy(h_output, d_output, M * sizeof(float), hipMemcpyDeviceToHost);
    stopTime(&timer);
    printElapsedTime(timer, "Device to Host Memcpy");

    // 验证结果
    exclusive_scan_cpu(h_input, h_reference, M);

    bool correct = true;
    for (unsigned int i = 0; i < M; ++i) {
        if (fabs(h_output[i] - h_reference[i]) > 1e-3f) {
            std::cerr << "Mismatch at index " << i << ": GPU = "
                      << h_output[i] << ", CPU = " << h_reference[i] << std::endl;
            correct = false;
            break;
        }
    }

    std::cout << (correct ? "Scan result correct ✅" : "Scan result incorrect ❌") << std::endl;

    // 清理资源
    delete[] h_input;
    delete[] h_output;
    delete[] h_reference;
    hipFree(d_input);
    hipFree(d_output);

    return 0;
}