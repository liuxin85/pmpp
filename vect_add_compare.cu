
#include <hip/hip_runtime.h>

void vecadd_cpu(float* x, float*y, float *z, int N){
    for(unsigned int i  = 0; i < N; ++i){
        z[i] = x[i] +  y[i];
    }
}
__global__ void vecadd_kernel (float* x, float* y, float* z, int N){
    unsigned int i = blockDim.x * blockIdx.x  + threadIdx.x;
    if(i < N){
      z[i] = x[i] + y[i];
    }
}

void vecadd_gpu(float* x , float* y, float*z, int N){
    // Allocate GPU memory
    float *x_d, *y_d, *z_d;
    hipMalloc((void**)&x_d, N * sizeof(float));
    hipMalloc((void**)&y_d, N * sizeof(float));
    hipMalloc((void**)&z_d, N * sizeof(float));

    // copy to the GPU
    hipMemcpy(x_d, x, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(y_d, y, N*sizeof(float), hipMemcpyHostToDevice);


    // Call a cpu kernel function  (lunch a grid of threads)
    const unsigned int numThreadsPerBlock  = 512;
    const unsigned int numBlocks = (numThreadsPerBlock +  N -1 )/ numThreadsPerBlock;
    vecadd_kernel<<<numBlocks,numThreadsPerBlock>>>(x_d, y_d, z_d, N);
    hipError_t err = hipDeviceSynchronize();
    if(err != hipSuccess){

    }

    // copy from the GPU
    hipMemcpy(z, z_d, N*sizeof(float), hipMemcpyDeviceToHost);

    // Deallocate GPU memory
    hipFree(x_d);
    hipFree(y_d);
    hipFree(z_d);
}

int main(int argc, char** argv){
    hipDeviceSynchronize();


    unsigned int N = (argc > 1) ? (atoi(argv[1])) : (1 << 25);
    float* x = (float*) malloc(N*sizeof(float));
    float* y = (float*) malloc(N* sizeof(float));
    float* z = (float*) malloc(N* sizeof(float));

    for(unsigned int i = 0; i< N; ++i){
        x[i] = rand();
        y[i] = rand();
    }
    // vector addtion on cpu
    vecadd_cpu(x, y, z, N);


    // vector addtion on GPU
    vecadd_gpu(x,y,z,N);

    free(x);
    free(y);
    free(z);

    return 0;
}